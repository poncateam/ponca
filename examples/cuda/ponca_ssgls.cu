#include "hip/hip_runtime.h"
/*
This Source Code Form is subject to the terms of the Mozilla Public
License, v. 2.0. If a copy of the MPL was not distributed with this
file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

/*!
\file examples/Ponca/ssgls.cu
\brief Screen space GLS using c++/CUDA
*/

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <vector>
#include <chrono>

#include <png.h>

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int

#include <Ponca/src/Fitting/basket.h>
#include <Ponca/src/Fitting/gls.h>
#include <Ponca/src/Fitting/orientedSphereFit.h>
#include <Ponca/src/Fitting/weightFunc.h>
#include <Ponca/src/Fitting/weightKernel.h>



/**************************************************************************************************/
/* IO (source: http://zarb.org/~gc/html/libpng.html )                                             */
/**************************************************************************************************/

class PNGImage
{
public:
  inline bool load(const char *file_name);
  inline bool loaded () const { return ! row_pointers.empty(); }
  inline bool save(const char *file_name);

  inline png_uint_32 width()  const { return m_width; };
  inline png_uint_32 height() const { return m_height; };

  inline const std::vector<png_bytep>& buffer() const { return row_pointers; }
  inline std::vector<png_bytep>& buffer() { return row_pointers; }
  inline png_byte colorType() const { return png_get_color_type(png_ptr, info_ptr);}

  ~PNGImage() { for (auto e: row_pointers) delete e; row_pointers.clear(); }
private:
  png_uint_32 m_width, m_height;
  png_byte color_type;
  png_byte bit_depth;

  png_structp png_ptr;
  png_infop info_ptr;
  int number_of_passes;
  std::vector<png_bytep> row_pointers;

  using vecSizeT = typename std::vector<png_bytep>::size_type;
};

bool
PNGImage::load(const char* file_name)
{
    unsigned char header[8];    // 8 is the maximum size that can be checked

    /* open file and test for it being a png */
    FILE *fp = fopen(file_name, "rb");
    if (!fp)
    {
        std::cerr << "[read_png_file] File " \
                  <<  file_name
                  << " could not be opened for reading"
                  << std::endl;
        return false;
    }

    fread(header, 1, 8, fp);
    if (png_sig_cmp(header, 0, 8))
    {
        std::cerr << "[read_png_file] File " \
                  <<  file_name
                  << " is not recognized as a PNG file"
                  << std::endl;
        return false;
    }


    /* initialize stuff */
    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, nullptr, nullptr, nullptr);

    if (!png_ptr)
    {
        std::cerr << "[read_png_file] png_create_read_struct failed"
                  << std::endl;
        return false;
    }

    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr)
    {
        std::cerr << "[read_png_file] png_create_info_struct failed"
                  << std::endl;
        return false;
    }

    if (setjmp(png_jmpbuf(png_ptr)))
    {
        std::cerr << "[read_png_file] Error during init_iod"
                  << std::endl;
        return false;
    }

    png_init_io(png_ptr, fp);
    png_set_sig_bytes(png_ptr, 8);

    png_read_info(png_ptr, info_ptr);

    m_width = png_get_image_width(png_ptr, info_ptr);
    m_height = png_get_image_height(png_ptr, info_ptr);
    color_type = png_get_color_type(png_ptr, info_ptr);
    bit_depth = png_get_bit_depth(png_ptr, info_ptr);

    number_of_passes = png_set_interlace_handling(png_ptr);
    png_read_update_info(png_ptr, info_ptr);


    /* read file */
    if (setjmp(png_jmpbuf(png_ptr)))
    {
        std::cerr << "[read_png_file] Error during read_image"
                  << std::endl;
        return false;
    }

    row_pointers.resize( m_height );
    for (vecSizeT y=0; y< vecSizeT(m_height); y++)
      row_pointers[y] = (png_byte*) (malloc(png_get_rowbytes(png_ptr,info_ptr)));

    png_read_image(png_ptr, row_pointers.data());

    fclose(fp);

    return true;
}

bool
PNGImage::save(const char* file_name) {
  /* create file */
  FILE *fp = fopen(file_name, "wb");
  if (!fp)
  {
      std::cerr << "[write_png_file] File " \
                <<  file_name
                << " could not be opened for reading"
                << std::endl;
      return false;
  }


  /* initialize stuff */
  png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, nullptr, nullptr, nullptr);

  if (!png_ptr)
  {
      std::cerr << "[write_png_file] png_create_write_struct failed"
                << std::endl;
      return false;
  }

  info_ptr = png_create_info_struct(png_ptr);
  if (!info_ptr)
  {
      std::cerr << "[write_png_file] png_create_info_struct failed"
                << std::endl;
      return false;
  }

  if (setjmp(png_jmpbuf(png_ptr)))
  {
      std::cerr << "[write_png_file] Error during init_io"
                << std::endl;
      return false;
  }

  png_init_io(png_ptr, fp);


  /* write header */
  if (setjmp(png_jmpbuf(png_ptr)))
  {
      std::cerr << "[write_png_file] Error during writing header"
                << std::endl;
      return false;
  }

  png_set_IHDR(png_ptr, info_ptr, m_width, m_height,
               bit_depth, color_type, PNG_INTERLACE_NONE,
               PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);

  png_write_info(png_ptr, info_ptr);


  /* write bytes */
  if (setjmp(png_jmpbuf(png_ptr)))
  {
      std::cerr << "[write_png_file] Error during writing bytes"
                << std::endl;
      return false;
  }

  png_write_image(png_ptr, row_pointers.data());


  /* end write */
  if (setjmp(png_jmpbuf(png_ptr)))
  {
      std::cerr << "[write_png_file] Error during end of write"
                << std::endl;
      return false;
  }


  png_write_end(png_ptr, nullptr);

  fclose(fp);
  return true;
}

/**************************************************************************************************/
/* Ponca initialization                                                                           */
/**************************************************************************************************/
//! [mypoint]
class ScreenSpacePoint
{
public:
    enum {Dim = 3};
    typedef float Scalar;
    typedef Eigen::Matrix<Scalar, Dim, 1>   VectorType;
    typedef Eigen::Matrix<Scalar, 2,   1>   ScreenVectorType;
    typedef Eigen::Matrix<Scalar, Dim, Dim> MatrixType;

    PONCA_MULTIARCH inline ScreenSpacePoint(const VectorType       &_pos    = VectorType::Zero(),
                                      const VectorType       &_normal = VectorType::Zero(),
                                      const ScreenVectorType &_spos   = ScreenVectorType::Zero())
        : m_pos(_pos), m_normal(_normal), m_spos(_spos){}

    PONCA_MULTIARCH inline const VectorType& pos()	const { return m_pos; }
    PONCA_MULTIARCH inline const VectorType& normal()	const { return m_normal; }
    PONCA_MULTIARCH inline const ScreenVectorType& spos() const { return m_spos; }

    PONCA_MULTIARCH inline VectorType& pos()	 { return m_pos; }
    PONCA_MULTIARCH inline VectorType& normal()	 { return m_normal; }
    PONCA_MULTIARCH inline ScreenVectorType& spos() { return m_spos; }

private:
    VectorType	m_pos, m_normal;
    ScreenVectorType m_spos;
};
//! [mypoint]

typedef ScreenSpacePoint::Scalar Scalar;
typedef ScreenSpacePoint::VectorType VectorType;
typedef ScreenSpacePoint::ScreenVectorType ScreenVectorType;

//! [w_def]
class ProjectedWeightFunc: public Ponca::DistWeightFunc<ScreenSpacePoint,Ponca::SmoothWeightKernel<Scalar> >
{
public:
    typedef ScreenSpacePoint::Scalar Scalar;
    typedef ScreenSpacePoint::VectorType VectorType;
    using Base = Ponca::DistWeightFunc<ScreenSpacePoint,Ponca::SmoothWeightKernel<Scalar> >;

    PONCA_MULTIARCH inline ProjectedWeightFunc(const VectorType& _evalPos = VectorType::Zero(), const Scalar& _t = Scalar(1.), const Scalar _dz = 0.f)
        : Base(_evalPos, _t), m_dz(_dz) {}

    PONCA_MULTIARCH inline Base::WeightReturnType w(const VectorType& _relativePos, const ScreenSpacePoint&  _attributes) const
    {
        PONCA_MULTIARCH_STD_MATH(abs);
        Scalar d  = _attributes.spos().norm();
        const float dz = abs(_relativePos[2]);
        if (d > m_t || (m_dz != Scalar(0) && dz > m_dz))
        {
            return {Scalar(0.), _relativePos};
        }
        return {m_wk.f(d/m_t), _relativePos};
    }
private:
    float m_dz;
};
//! [w_def]

//! [fit_def]
typedef Ponca::Basket< ScreenSpacePoint,
                           ProjectedWeightFunc,
                           Ponca::OrientedSphereFit,
                           Ponca::GLSParam> ScreenSpaceFit;
//! [fit_def]

//! [data_acces]
__device__ int getId(const int _x,
                     const int _y,
                     const int _width,
                     const int _height,
                     const int _component,
                     const int _nbComponent)
{
    return (_component) + _nbComponent*(_x + _y * _width);
}

__device__ VectorType getVector(const int _x,
                                const int _y,
                                const int _width,
                                const int _height,
                                const float* _buffer)
{
    VectorType r;
    r << Scalar(_buffer[getId(_x,_y,_width,_height,0,3)]),
        Scalar(_buffer[getId(_x,_y,_width,_height,1,3)]),
        Scalar(_buffer[getId(_x,_y,_width,_height,2,3)]);
    return r;
}
//! [data_acces]

//! [kernel]
__global__ void doGLS_kernel( int _imgw, int _imgh, int _scale,
            float _maxDepthDiff, float* _positions, float* _normals,
            float* _result)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x * bw + tx;
    int y = blockIdx.y * bh + ty;

    int idx = y * _imgw + x;

    if((x >= _imgw || y >= _imgh))
    {
            return;
    }
    else if(getVector(x, y, _imgw, _imgh, _normals).squaredNorm() == 0.f)
    {
    _result[idx] = 0.f;
            return;
    }

    VectorType one = VectorType::Ones();
    const float scale2 = float(_scale * _scale);

//    VectorType vvvvv = getVector(x, y, _imgw, _imgh, _positions);
//    VectorType nnnnn = getVector(x, y, _imgw, _imgh, _normals);
//    _result[idx] = vvvvv(2);
//    return;

    ScreenSpaceFit fit;
    fit.init();
    fit.setNeighborFilter({
        getVector(x, y, _imgw, _imgh, _positions) * 2.f - one,
        float(_scale),
        _maxDepthDiff
    });

    _result[idx] = 0.f;

    // collect neighborhood
    for(int dy = -_scale; dy != _scale + 1; dy++)
    {
        for(int dx = -_scale; dx != _scale + 1; dx++)
        {
            float dist2 = dy*dy + dx*dx;
            // Check if we are in the circular screen-space neighborhood
            if (dist2 < scale2)
            {
                int nx, ny; // neighbor ids

    nx = x + dx;
    ny = y + dy;

    // Check image boundaries
    if(nx >= 0 && ny >= 0 && nx < _imgw && ny < _imgh)
    {
        ScreenSpacePoint::VectorType n = getVector(nx, ny, _imgw, _imgh, _normals);
        // add nei only when the normal is properly defined
        if(n.squaredNorm() != 0.f)
        {
      // RGB to XYZ remapping
      n =  2.f * n - one;
      n.normalize();

      ScreenSpacePoint::ScreenVectorType xyCoord;
      xyCoord[0] = dx;
      xyCoord[1] = dy;

      ScreenSpacePoint::VectorType p = getVector(nx, ny, _imgw, _imgh, _positions) * 2.f - one;
      // GLS computation
      fit.addNeighbor(ScreenSpacePoint(p, n, xyCoord));
        }
    }
      }
  }
    }

    // closed form minimization
    fit.finalize();
    _result[idx] = fit.kappa();
}
//! [kernel]

/**
* \brief RGB basic color representation
*/
typedef struct
{
    double r,g,b;
}Color;

/**
* \brief Return Color corresponding to the _value param. Simulating a "seismic" like color map
*/
__host__ Color getColor(float _value, float _valueMin, float _valueMax)
{
    Color c = {1.0, 1.0, 1.0};
    double dv;

    // Unknown values in our kernel
    if(_value == 0.)
    {
        return c;
    }

    // Threshold
    if (_value < _valueMin)
    {
        _value = _valueMin;
    }

    if (_value > _valueMax)
    {
        _value = _valueMax;
    }

    // Interval
    dv = _valueMax - _valueMin;

    // Seismic color map like
    if(_value < (_valueMin + 0.5 * dv))
    {
        c.r = 2 * (_value - _valueMin) / dv;
        c.g = 2 * (_value - _valueMin) / dv;
        c.b = 1;
    }
    else
    {
        c.b = 2 - 2 * (_value - _valueMin) / dv;
        c.g = 2 - 2 * (_value - _valueMin) / dv;
        c.r = 1;
    }

    return c;
}

/**
* \brief Init input datas to be used on host
*/
__host__ bool initInputDatas(const PNGImage& positions, const PNGImage& normals,
                             std::vector<float>& positionsInfos,
                             std::vector<float>& normalsInfos,
                             unsigned int& width, unsigned int& height)
{

    if (positions.colorType() != PNG_COLOR_TYPE_RGB) {
      std::cerr << "[process_file] color_type of input file must be PNG_COLOR_TYPE_RGB ("
                << PNG_COLOR_TYPE_RGB
                << ") (is "
                << positions.colorType()
                << ")"
                << std::endl;
      return false;
    }


    width = positions.width();
    height = positions.height();

    positionsInfos.resize(width*height*3);
    normalsInfos.resize(width*height*3);

    auto pbuf = positions.buffer();
    auto nbuf = normals.buffer();

    for (int j = 0; j < height; ++j) {
        png_bytep pcol = pbuf[j];
        png_bytep ncol = nbuf[j];

        float* pout = positionsInfos.data()+j*width*3;
        float* nout = normalsInfos.data()+j*width*3;

        auto scaleValues = [](const png_byte& in){ return in / 255.f * 2.f - 1.f; };
        std::transform(pcol, pcol+width*3, pout, scaleValues );
        std::transform(ncol, ncol+width*3, nout, scaleValues );
    }

    return true;
}

/**
* \brief Save _results into png image
*/
__host__ bool saveResult(float* _results,
                         const char* _positionsFilename, const char* _resultFilename)
{

    PNGImage result;
    if(!result.load(_positionsFilename))
    {
        fprintf(stderr, "Cannot load positions\n");
        return false;
    }

    int width = result.width();
    int height = result.height();

    auto pbuf = result.buffer().data();

    for (int j = 0; j < height; ++j) {
        float* pin = _results+j*width;
        png_bytep col = pbuf[j];
        for (int i = 0; i < width; ++i) {
            //check nan
            if(std::isnan(pin[i]))
            {
                pin[i] = 0.f;
            }
            Color c = getColor(pin[i], -10., 10.);

            col[i * 3 + 0] = c.r * 255.;
            col[i * 3 + 1] = c.g * 255.;
            col[i * 3 + 2] = c.b * 255.;
        }
    }

    if(!result.save(_resultFilename))
    {
        fprintf(stderr, "Cannot save image\n");
    }

    return true;
}

__host__ int adjust(int n, int blockSize)
{
   if (n < blockSize) { return n; }
   return (n / blockSize + (n % blockSize == 0 ? 0 : 1)) * blockSize;
}

int main()
{
    const char *positionsFilename = "./data/ssgls_sample_wc.png";
    const char *normalsFilename = "./data/ssgls_sample_normal.png";
    const char *resultFilename = "./ssgls_results.png";

    PNGImage positions, normals;

    if(!positions.load(positionsFilename) || ! normals.load(normalsFilename))
    {
        return 0;
    }

    float fScale = 10.f;
    float fMaxDepthDiff = 0.00f;
    unsigned int width = 0;
    unsigned int height = 0;
    std::vector<float> positionsInfos, normalsInfos;

    if(!initInputDatas(positions, normals, positionsInfos, normalsInfos, width, height))
    {
        return 0;
    }

    std::cout << "Image size : " << width << "*" << height << std::endl;

    /*********** Init Output ************/
    float *results = new float[width*height];
    std::fill( results, results + width*height, 0.f );

    /************* Init device mem *************/
    size_t sizeResults = width * height * sizeof(float);
    size_t sizeImg = width * height * 3 * sizeof(float);

    float* positionsInfos_device;
    float* normalsInfos_device;
    float* results_device;

    hipMalloc(&positionsInfos_device, sizeImg);
    hipMemcpy(positionsInfos_device, positionsInfos.data(), sizeImg, hipMemcpyHostToDevice);

    hipMalloc(&normalsInfos_device, sizeImg);
    hipMemcpy(normalsInfos_device, normalsInfos.data(), sizeImg, hipMemcpyHostToDevice);

    hipMalloc(&results_device, sizeResults);
    hipMemcpy(results_device, results, sizeResults, hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();
    /************* Memory conf *************/

    // calculate grid size
    dim3 block(32, 32, 1);
    dim3 grid(adjust(width, block.x) / block.x, adjust(height, block.y) / block.y, 1);

    /************* Kernel Call *************/

    std::cout << "ssCurvature running..." << std::endl;

    // dry run: first call is always slower
    doGLS_kernel<<<grid, block>>>(width, height, fScale, fMaxDepthDiff, positionsInfos_device, normalsInfos_device, results_device);

    int nbrun = 100;
    auto start = std::chrono::system_clock::now();
    for( int i = 0; i != nbrun; ++i) {
      doGLS_kernel<<<grid, block>>>(width, height, fScale, fMaxDepthDiff, positionsInfos_device, normalsInfos_device, results_device);
      hipDeviceSynchronize();	// Wait for the GPU launched work to complete
    }
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = (end-start)/double(nbrun);

    err = hipGetLastError();

    std::cout << "ssCurvature completed in " << diff.count() << " s" << std::endl;

    /************* Get Results *************/
    hipMemcpy(results, results_device, sizeResults, hipMemcpyDeviceToHost);

    err = hipGetLastError();

    std::cout << "Finalizing..." << std::endl;

    /********** Cuda Free ************/
    hipFree(positionsInfos_device);
    hipFree(normalsInfos_device);
    hipFree(results_device);

    err = hipGetLastError();

    /********** Saving _result ************/
    if(!saveResult(results, positionsFilename, resultFilename))
    {
        return 0;
    }

    /********** Free Memory *********/
    delete [] results;

    hipDeviceReset();
    err = hipGetLastError();

    std::cout << "Finished !" << std::endl;

    return 0;
}

